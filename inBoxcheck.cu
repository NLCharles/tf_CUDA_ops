
#include <hip/hip_runtime.h>
// This is the naive implementation of in box check with all matrices squeezed to vector
__global__ void inBoxKernel(const float *A, const float *B, int *C, int numElements){
    int i = (blockDim.x * blockIdx.x + threadIdx.x)*2;
    float t11;
    float t12;
    float t21;
    float t22;
    if (i/2 < numElements)
    {
        if ((A[i]<B[0] || (A[i] >B[4]))||((A[i+1]>B[3]) || (A[i+1] <B[7])))
            {
                C[i/2]=-1;
            }
        else
        {
           t11=(A[i]-B[0])*(B[2]-B[0])+(A[i+1]-B[1])*(B[3]-B[1]);
           t12=(A[i]-B[4])*(B[6]-B[4])+(A[i+1]-B[5])*(B[7]-B[5]);
           t21=(A[i]-B[2])*(B[4]-B[2])+(A[i+1]-B[3])*(B[5]-B[3]);
           t22=(A[i]-B[6])*(B[2]-B[6])+(A[i+1]-B[7])*(B[1]-B[7]);
           if (t11>=1e-20 && t12>=1e-20 && t21>=1e-20 && t22>=1e-20) 
           {
                C[i/2]=1;
           }
           else
           {
                C[i/2]=-1;
           }
        }
    }
}

void inBoxLauncher(const float *A, const float *B, int *C, int numElements){
    int threadsPerBlock = 512;
    int blocksPerGrid =(numElements - 1) / threadsPerBlock +1;
    inBoxKernel<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, numElements);
}


